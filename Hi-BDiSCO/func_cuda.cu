#include "hip/hip_runtime.h"
#include "func_cuda.h"

double* d_D;
double* d_Chol;

//Device Matrices
int* d_type;
double* d_r;
double* d_a;
double* d_b;
double* d_c;
double* d_alpha;
double* d_beta;
double* d_gamma;
double* d_length;
double* d_a_dna;
double* d_b_dna;
double* d_c_dna;
double* d_alpha_p;
double* d_beta_p;
double* d_gamma_p;
double* d_phi_o;
double* d_force;
double* d_torque;
double* d_Energy;
double* d_core_pos;
double* d_core_q;
double* d_d_theta;
double* d_rd;

double* d_r_n;
double* d_a_n;
double* d_b_n;
double* d_c_n;
double* d_alpha_n;
double* d_beta_n;
double* d_gamma_n;
double* d_length_n;
double* d_a_dna_n;
double* d_b_dna_n;
double* d_c_dna_n;
double* d_alpha_p_n;
double* d_beta_p_n;
double* d_gamma_p_n;
double* d_force_n;
double* d_torque_n;

double* d_force_tmp;
double* d_torque_tmp;

double* d_force_m;
double* d_torque_m;

double* d_rad_all;
double* d_r_all;

double* d_Energy_m;

int* d_ex_force_m;

__device__ void first_coord_cuda(int t, double* r, double* a, double* b, double* c, double* r_f){

        double b_m[3];
        double si, co;

        si = sin(theta);
        co = cos(theta);

        if (t==0){
                r_f[0] = r[0];
                r_f[1] = r[1];
                r_f[2] = r[2];
        }else{
                b_m[0] = -si*a[0]+co*b[0];
                b_m[1] = -si*a[1]+co*b[1];
                b_m[2] = -si*a[2]+co*b[2];

                r_f[0] = r[0] - ro*b_m[0]+d1*c[0];
                r_f[1] = r[1] - ro*b_m[1]+d1*c[1];
                r_f[2] = r[2] - ro*b_m[2]+d1*c[2];
        }

}

__device__ void second_coord_cuda(int t, double* r, double* a, double* b, double* c, double* r_s){

        if (t==0){
                r_s[0] = r[0];
                r_s[1] = r[1];
                r_s[2] = r[2];
        }else{
                r_s[0] = r[0] - (ro*b[0]+d1*c[0]);
                r_s[1] = r[1] - (ro*b[1]+d1*c[1]);
                r_s[2] = r[2] - (ro*b[2]+d1*c[2]);
        }

}

__device__ void norm_cuda(double* r, double& result){
    int i;
    double norma = 0;
    for (i = 0; i<3; i++){
        norma = norma + r[i]*r[i];
    }
    result = sqrt(norma);
}

__device__ void cross_product_cuda(double* r1, double* r2, double* product){

    product[0] = r1[1] * r2[2] - r1[2] * r2[1];
    product[1] = -1 * (r1[0] * r2[2] - r1[2] * r2[0]);
    product[2] = r1[0] * r2[1] - r1[1] * r2[0];

}


__device__ void rotate_cuda(int n, int n3, double* a, double* b, double* c, double* a_n, double* b_n, double* c_n, double* d_theta, double dt){

        double wa, wb, wc, g1, g2, g3;
        double z, z2, wa2, wb2, wc2, czt, Omczt, szt;
        int i, i1, i2, i3;

        for (i = 0; i < n; i++){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                wa = d_theta[i1];
                wb = d_theta[i2];
                wc = d_theta[i3];
                wa2 = wa*wa;
                wb2 = wb*wb;
                wc2 = wc*wc;
                z2 = wa2 + wb2 + wc2;
                z = sqrt(z2);
                czt = cos(z*dt);
                szt = sin(z*dt);
                Omczt = 1.0 - czt;

                if (z2 > 0.0){
                        //rotation of a
                        g1 = ((wb2+wc2)*czt+wa2)/z2;
                        g2 = wa*wb*Omczt/z2 + wc*szt/z;
                        g3 = wa*wc*Omczt/z2 - wb*szt/z;
                        a_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        a_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        a_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                        //rotation of b
                        g1 = wa*wb*Omczt/z2 - wc*szt/z;
                        g2 = ((wa2+wc2)*czt+wb2)/z2;
                        g3 = wb*wc*Omczt/z2 + wa*szt/z;
                        b_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        b_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        b_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                        //rotation of c
                        g1 = wa*wc*Omczt/z2 + wb*szt/z;
                        g2 = wb*wc*Omczt/z2 - wa*szt/z;
                        g3 = ((wa2+wb2)*czt+wc2)/z2;
                        c_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        c_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        c_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                }else{
                        a_n[i1] = a[i1];
                        a_n[i2] = a[i2];
                        a_n[i3] = a[i3];
                        b_n[i1] = b[i1];
                        b_n[i2] = b[i2];
                        b_n[i3] = b[i3];
                        c_n[i1] = c[i1];
                        c_n[i2] = c[i2];
                        c_n[i3] = c[i3];
                }

        }

}

__device__ void Force_Stretching_cuda(double h, double* coord1, double* coord2, double lo, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;

        double distance = 0.0;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

        distance = sqrt(distance);

        Energy = Energy + h*(distance-lo)*(distance-lo)/2;

        force = -h*(distance-lo);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
//		if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
//                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
//                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
//                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void Force_Bending_cuda(double g, double beta, double beta_o, double* coord1, double* coord2, double* coord3, double* force_projection1, double* force_projection2, double* force_projection3, double& Energy){

        double force = 0.0;

        double distance1, distance2;
        double norm_ri, norm_rk;

        double ji[3], jk[3], kj[3], ri[3], rk[3], product[3];

        Energy = Energy + g*(beta-beta_o)*(beta-beta_o)/2;

        force = -g*(beta-beta_o);

        distance1 = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);
        distance2 = (coord2[0]-coord3[0])*(coord2[0]-coord3[0]) + (coord2[1]-coord3[1])*(coord2[1]-coord3[1]) + (coord2[2]-coord3[2])*(coord2[2]-coord3[2]);

        distance1 = sqrt(distance1);
        distance2 = sqrt(distance2);

        ji[0] = coord2[0]-coord1[0];
        ji[1] = coord2[1]-coord1[1];
        ji[2] = coord2[2]-coord1[2];
        jk[0] = coord2[0]-coord3[0];
        jk[1] = coord2[1]-coord3[1];
        jk[2] = coord2[2]-coord3[2];
        kj[0] = coord3[0]-coord2[0];
        kj[1] = coord3[1]-coord2[1];
        kj[2] = coord3[2]-coord2[2];

        cross_product_cuda(ji, jk, product);
        for (int i = 0; i < 3; i++){
                ri[i] = product[i];
        }
        cross_product_cuda(ji, ri, product);
        for (int i = 0; i < 3; i++){
                ri[i] = product[i];
        }

        norm_cuda(ri, norm_ri);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = (force/distance1)*ri[i]/norm_ri;
                force_projection2[i] = -(force/distance1)*ri[i]/norm_ri;
        }

        cross_product_cuda(ji, jk, product);
        for (int i = 0; i < 3; i++){
                rk[i] = product[i];
        }
        cross_product_cuda(kj, rk, product);
        for (int i = 0; i < 3; i++){
                rk[i] = product[i];
        }

        norm_cuda(rk, norm_rk);

        for (int j = 0; j < 3; j++){
                force_projection2[j] = force_projection2[j] - (force/distance2)*rk[j]/norm_rk;
                force_projection3[j] = (force/distance2)*rk[j]/norm_rk;
        }

}


__device__ void Bending_force_projection_cuda(double g, double beta, double beta_b, double length, double* a_f, double* a_b, double* a, double* force_projection1, double* force_projection2, double& Energy){

        double Ai[3], Bi[3];
        double c1, c2, g1, g2;

        Energy = Energy + g*(beta-beta_b)*(beta-beta_b)/2;

        if (beta >= 1e-10){
                g1 = beta/(sin(beta)*length);
        }else{
                g1 = 1.0/length;
        }
        c1 = cos(beta);

        Ai[0] = g1*(a_f[0]-c1*a[0]);
        Ai[1] = g1*(a_f[1]-c1*a[1]);
        Ai[2] = g1*(a_f[2]-c1*a[2]);

        if (beta_b >= 1e-10){
                g2 = beta_b/(sin(beta_b)*length);
        }else{
                g2 = 1.0/length;
        }
        c2 = cos(beta_b);

        Bi[0] = g2*(a_b[0]-c2*a[0]);
        Bi[1] = g2*(a_b[1]-c2*a[1]);
        Bi[2] = g2*(a_b[2]-c2*a[2]);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = -g*(Ai[i]+Bi[i]);
                force_projection2[i] = g*(Ai[i]+Bi[i]);
        }

}

__device__ void Twisting_force_projection_cuda(double s, double alpha, double beta, double gamma, double phi_o, double length, double alpha_b, double beta_b, double gamma_b, double phi_o_b, double gamma_n, double* b, double* c, double* force_projection1, double* force_projection2, double& Energy){

        double Chi[3], Zhi[3];
        double g1, g2, c1, c2, s1, s2;

        Energy = Energy + s*((alpha+gamma-phi_o)*(alpha+gamma-phi_o))/2;

        g1 = (alpha+gamma-phi_o)*tan(0.5*beta)/length;
        c1 = cos(alpha);
        s1 = sin(alpha);

        g2 = (alpha_b+gamma_b-phi_o_b)*tan(0.5*beta_b)/length;
        c2 = cos(gamma_n);
        s2 = sin(gamma_n);

        Chi[0] = g1*(c1*c[0]-s1*b[0]);
        Chi[1] = g1*(c1*c[1]-s1*b[1]);
        Chi[2] = g1*(c1*c[2]-s1*b[2]);
        Zhi[0] = g2*(c2*c[0]+s2*b[0]);
        Zhi[1] = g2*(c2*c[1]+s2*b[1]);
        Zhi[2] = g2*(c2*c[2]+s2*b[2]);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = s*(Chi[i]+Zhi[i]);
                force_projection2[i] = -s*(Chi[i]+Zhi[i]);
        }
}

__device__ void Force_Electrostatics_cuda(double q1, double q2, double epslon, double kappa, double* coord1, double* coord2, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;
        double Rcut = 7.0;
        double distance = 0.0;
        double temp;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

        distance = sqrt(distance);

        if (distance < Rcut){
                temp = -kappa*distance;
                force = ((q1*q2*(kappa*distance+1))/(4*PI*epslon*distance*distance))*exp(temp);
                Energy = Energy + (q1*q2/(4*PI*epslon*distance))*exp(temp);
        }

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
                if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void Force_Exclude_Volume_cuda(double k_ev, double sigma, double* coord1, double* coord2, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;
        double vdw_cut = 4.0;
        double distance = 0.0;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

        distance = sqrt(distance);

        if (distance < vdw_cut){
                force = -k_ev*((6*pow(sigma,6))/pow(distance,7) - (12*pow(sigma,12))/pow(distance,13));
                Energy = Energy + k_ev*(pow(sigma/distance,12) - pow(sigma/distance,6));
        }

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
                if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void torque_due_to_force_cuda(double* force, double* coord_f, double* coord_c, double* a, double* b, double* c, double* torque){

        //Given exact coordinates of the bead with force applied (coord_f) and the exact coordinate of the center bead (coord_c)

        double fa, fb, fc;
        double comp[3];

        comp[0] = (coord_f[0]-coord_c[0])*a[0] + (coord_f[1]-coord_c[1])*a[1] + (coord_f[2]-coord_c[2])*a[2];
        comp[1] = (coord_f[0]-coord_c[0])*b[0] + (coord_f[1]-coord_c[1])*b[1] + (coord_f[2]-coord_c[2])*b[2];
        comp[2] = (coord_f[0]-coord_c[0])*c[0] + (coord_f[1]-coord_c[1])*c[1] + (coord_f[2]-coord_c[2])*c[2];


        fa = a[0]*force[0] + a[1]*force[1] + a[2]*force[2];
        fb = b[0]*force[0] + b[1]*force[1] + b[2]*force[2];
        fc = c[0]*force[0] + c[1]*force[1] + c[2]*force[2];

        torque[0] = fc*comp[1] - fb*comp[2];
        torque[1] = fa*comp[2] - fc*comp[0];
        torque[2] = fb*comp[0] - fa*comp[1];

}

__device__ void torque_due_to_force_relative_cuda(double* force, double* comp, double* a, double* b, double* c, double* torque){

        //Given relative position (comp) of the bead with force applied

        double fa, fb, fc;

        fa = a[0]*force[0] + a[1]*force[1] + a[2]*force[2];
        fb = b[0]*force[0] + b[1]*force[1] + b[2]*force[2];
        fc = c[0]*force[0] + c[1]*force[1] + c[2]*force[2];

        torque[0] = fc*comp[1] - fb*comp[2];
        torque[1] = fa*comp[2] - fc*comp[0];
        torque[2] = fb*comp[0] - fa*comp[1];

}

__global__ void Diffusion_Tensor_CUDA(int n, int n3, double* r, double a1, double a2, double* rad, double* D, double* Chol){

        // Calculate Row and Column
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int column = blockIdx.x * blockDim.x + threadIdx.x;

        // indexing

        double sij[3];
        double ssq, s, f, f1, f2;
        int i, j, k, l;

        // Diffusion Tensor

        i = row / 3;
        j = column / 3;
        k = row % 3;
        l = column % 3;

        if (row < n3 and column < n3){

                if (row == column){
                        D[row*n3+column] = a1/rad[i];
                        Chol[row*n3+column] = a1/rad[i];
                } else if (i==j and k!=l){
                        D[row*n3+column] = 0;
                        Chol[row*n3+column] = 0;
                } else {
                        sij[0] = r[3*i] - r[3*j];
                        sij[1] = r[3*i+1] - r[3*j+1];
                        sij[2] = r[3*i+2] - r[3*j+2];
                        ssq = sij[0]*sij[0] + sij[1]*sij[1] + sij[2]*sij[2];
                        s = sqrt(ssq);

                        if (s >= rad[i]+rad[j]){
                                f = (rad[i]*rad[i]+rad[j]*rad[j])/ssq;
                                f1 = 1.0 + f/3;
                                f2 = 1.0 -f;
                                if (k==l){
                                        D[row*n3+column] = (a2/s)*(f1+f2*sij[k]*sij[k]/ssq);
                                        Chol[row*n3+column] = (a2/s)*(f1+f2*sij[k]*sij[k]/ssq);
                                } else {
                                        D[row*n3+column] = (a2/s)*(f2*sij[k]*sij[l]/ssq);
                                        Chol[row*n3+column] = (a2/s)*(f2*sij[k]*sij[l]/ssq);
                                }

                        }else{
                                s = pow((rad[i]*rad[i]*rad[i]+rad[j]*rad[j]*rad[j])/2.0, 1.0/3);
                                if (k==l){
                                        D[row*n3+column] = (a1/s)*(1.0-9.0*sqrt(ssq)/(32*s) + 3.0*sij[k]*sij[k]/(32.0*s*sqrt(ssq)));
                                        Chol[row*n3+column] = (a1/s)*(1.0-9.0*sqrt(ssq)/(32*s) + 3.0*sij[k]*sij[k]/(32.0*s*sqrt(ssq)));
                                } else {
                                        D[row*n3+column] = (a1/s)*(3.0*sij[k]*sij[l]/(32.0*s*sqrt(ssq)));
                                        Chol[row*n3+column] = (a1/s)*(3.0*sij[k]*sij[l]/(32.0*s*sqrt(ssq)));
                                }
                        }
                }

        }
}

__global__ void Cholesky_Decomposition_mod(int n3, double* Chol){

        // Calculate Row and Column
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int column = blockIdx.x * blockDim.x + threadIdx.x;


        if (row < n3 and column < n3){
                if (row < column){
                        Chol[row*n3+column] = 0.0;
                }
                if (isnan(Chol[row*n3+column])){
                        Chol[row*n3+column] = 0.0;
                }
        }

}

__global__ void rd_cal(int n3, double* rd, double* Chol, double* p, double s2dt){

        int column = blockIdx.x * blockDim.x + threadIdx.x;

        if (column < n3){

                rd[column] = 0.0;

                for (int k = 0; k <= column; k++){
                        rd[column] += s2dt*Chol[column*n3+k]*p[k];
                }

        }
}

__global__ void translation_cal(int n_D3, int n3, double* r, double* r_n, double* rd, double del, double* force_global, double* D){

        int column = blockIdx.x * blockDim.x + threadIdx.x;

        int k;

        if (column < n_D3){

                if (column < n3){
                        r_n[column] = r[column] + rd[column];
                        for (k = 0; k < n3; k++){
                                r_n[column] = r_n[column] + del*D[column*n_D3+k]*force_global[k];
                        }
                }

	}

}

__global__ void rotation_cal(int n, double* d_theta, int* type, double time_step, double* torque, double* rr, double* a, double* b, double* c, double* a_n, double* b_n, double* c_n){

        int j = blockIdx.x * blockDim.x + threadIdx.x;

        int j1, j2, j3;

        if (j < n){
                j1 = 3*j;
                j2 = j1+1;
                j3 = j2+1;
                if (type[j] != 0){
                        d_theta[j1] = time_step*torque[j1]/(8*PI*eta*125.0) + rr[j1];
                        d_theta[j2] = time_step*torque[j2]/(8*PI*eta*125.0) + rr[j2];
                        d_theta[j3] = time_step*torque[j3]/(8*PI*eta*125.0) + rr[j3];
                }else{
                        d_theta[j1] = time_step*torque[j1]/(4*PI*eta*r_h*r_h*lo) + rr[j1];
                        d_theta[j2] = 0.0;
                        d_theta[j3] = 0.0;
                }
        }

        rotate_cuda(n, n*3, a, b, c, a_n, b_n, c_n, d_theta, 1.0);

}


__global__ void update_Euler_Angle_cuda(int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p){

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        double r_forw[3];
        double mi;
        double da[3], a_old[3];
        double a_m[3], b_m[3];
        double Ac, apg, f1, f2, ada, bda, si, co;
        double sa, ca, sb, cb, sg, cg;
        double R21, R22, R23, R31, R32, R33;
        int i1,i2,i3, if1,if2,if3, ic,ic1,ic2,ic3;
        int count;

	si = sin(theta);
        co = cos(theta);

	if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                if1 = i1+3;
                if2 = if1+1;
                if3 = if2+1;
	
		if (type[i] == 0){
                        a_old[0] = a[i1];
                        a_old[1] = a[i2];
                        a_old[2] = a[i3];
                        if (type[i+1] == 0){
                                r_forw[0] = r[if1] - r[i1];
                                r_forw[1] = r[if2] - r[i2];
                                r_forw[2] = r[if3] - r[i3];
                        }else{
                                b_m[0] = -si*a[if1] + co*b[if1];
                                b_m[1] = -si*a[if2] + co*b[if2];
                                b_m[2] = -si*a[if3] + co*b[if3];
                                r_forw[0] = r[if1] - ro*b_m[0] + d1*c[if1] - r[i1];
                                r_forw[1] = r[if2] - ro*b_m[1] + d1*c[if2] - r[i2];
                                r_forw[2] = r[if3] - ro*b_m[2] + d1*c[if3] - r[i3];
                        }
                        length[i] = r_forw[0]*r_forw[0] + r_forw[1]*r_forw[1] + r_forw[2]*r_forw[2];
                        length[i] = sqrt(length[i]);
                        mi = 1.0/length[i];
                        a[i1] = mi*r_forw[0];
                        a[i2] = mi*r_forw[1];
                        a[i3] = mi*r_forw[2];
			da[0] = a[i1] - a_old[0];
                        da[1] = a[i2] - a_old[1];
                        da[2] = a[i3] - a_old[2];

                        bda = b[i1]*da[0] +b[i2]*da[1] + b[i3]*da[2];
                        b[i1] = b[i1] - bda*a_old[0];
                        b[i2] = b[i2] - bda*a_old[1];
                        b[i3] = b[i3] - bda*a_old[2];

                        bda = b[i1]*a[i1] + b[i2]*a[i2] + b[i3]*a[i3];
                        b[i1] = b[i1] - bda*a[i1];
                        b[i2] = b[i2] - bda*a[i2];
                        b[i3] = b[i3] - bda*a[i3];

                        mi = 1.0/sqrt(b[i1]*b[i1] + b[i2]*b[i2] + b[i3]*b[i3]);
                        b[i1] = mi*b[i1];
                        b[i2] = mi*b[i2];
                        b[i3] = mi*b[i3];

                        c[i1] = a[i2]*b[i3] - a[i3]*b[i2];
                        c[i2] = a[i3]*b[i1] - a[i1]*b[i3];
                        c[i3] = a[i1]*b[i2] - a[i2]*b[i1];
                }else{
                        r_forw[0] = r[if1] - (r[i1]-ro*b[i1]-d1*c[i1]);
                        r_forw[1] = r[if2] - (r[i2]-ro*b[i2]-d1*c[i2]);
                        r_forw[2] = r[if3] - (r[i3]-ro*b[i3]-d1*c[i3]);
                        length[i] = r_forw[0]*r_forw[0] + r_forw[1]*r_forw[1] + r_forw[2]*r_forw[2];
                        length[i] = sqrt(length[i]);
                }
        }

	if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                if1 = i1+3;
                if2 = if1+1;
                if3 = if2+1;

                if (type[i]==0){
                        if(type[i+1]==0){
                                ada = a[i1]*a[if1]+a[i2]*a[if2]+a[i3]*a[if3];
                                if (ada > 1.0) ada = 1.0;
                                if (ada < -1.0) ada = -1.0;
                                beta[i] = acos(ada);
                                sb = sin(beta[i]);
                                if (beta[i] > 1e-10){
                                        f1 = (a[if1]*b[i1]+a[if2]*b[i2]+a[if3]*b[i3])/sb;
                                }else{
                                        f1 = (b[if1]*b[i1]+b[if2]*b[i2]+b[if3]*b[i3]);
                                }
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                Ac = acos(f1);
                                f2 = a[if1]*c[i1] + a[if2]*c[i2] + a[if3]*c[i3];
                                if (f2 >= 0){
                                        alpha[i] = Ac;
                                }else{
                                        alpha[i] = -Ac;
                                }

				f1 = (b[i1]*b[if1]+b[i2]*b[if2]+b[i3]*b[if3]+c[i1]*c[if1]+c[i2]*c[if2]+c[i3]*c[if3])/(1.0 + ada);
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                apg = acos(f1);
                                f2 = (c[i1]*b[if1]+c[i2]*b[if2]+c[i3]*b[if3]-(b[i1]*c[if1]+b[i2]*c[if2]+b[i3]*c[if3]))/(1.0 + ada);
                                if (f2 >= 0.0){
                                        gamma[i] = apg - alpha[i];
                                }else{
                                        gamma[i] = -apg - alpha[i];
                                }
                        }else{
                                a_m[0] = co*a[if1] + si*b[if1];
                                a_m[1] = co*a[if2] + si*b[if2];
                                a_m[2] = co*a[if3] + si*b[if3];
                                b_m[0] = -si*a[if1] + co*b[if1];
                                b_m[1] = -si*a[if2] + co*b[if2];
                                b_m[2] = -si*a[if3] + co*b[if3];

                                ada = a[i1]*a_m[0]+a[i2]*a_m[1]+a[i3]*a_m[2];
                                if (ada > 1.0) ada = 1.0;
                                if (ada < -1.0) ada = -1.0;
                                beta[i] = acos(ada);
                                sb = sin(beta[i]);
                                if (beta[i] > 1e-10){
                                        f1 = (a_m[0]*b[i1]+a_m[1]*b[i2]+a_m[2]*b[i3])/sb;
                                }else{
                                        f1 = (b_m[0]*b[i1]+b_m[1]*b[i2]+b_m[2]*b[i3]);
                                }
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                Ac = acos(f1);
                                f2 = a_m[0]*c[i1] + a_m[1]*c[i2] + a_m[2]*c[i3];
                                if (f2 >= 0){
                                        alpha[i] = Ac;
                                }else{
                                        alpha[i] = -Ac;
                                }

				f1 = (b[i1]*b_m[0]+b[i2]*b_m[1]+b[i3]*b_m[2]+c[i1]*c[if1]+c[i2]*c[if2]+c[i3]*c[if3])/(1.0 + ada);
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                apg = acos(f1);
                                f2 = (c[i1]*b_m[0]+c[i2]*b_m[1]+c[i3]*b_m[2]-(b[i1]*c[if1]+b[i2]*c[if2]+b[i3]*c[if3]))/(1.0 + ada);
                                if (f2 >= 0.0){
                                        gamma[i] = apg - alpha[i];
                                }else{
                                        gamma[i] = -apg - alpha[i];
                                }
                        }
                }else{
                        ic = 0;
                        for (count = 0; count <= i; count++){
                                if(type[count]==1) ic=ic+1;
                        }
                        ic = ic-1;
                        ic1 = 3*ic;
                        ic2 = ic1+1;
                        ic3 = ic2+1;

                        a_dna[ic1] = (r[if1] - (r[i1]-ro*b[i1]-d1*c[i1]))/length[i];
                        a_dna[ic2] = (r[if2] - (r[i2]-ro*b[i2]-d1*c[i2]))/length[i];
                        a_dna[ic3] = (r[if3] - (r[i3]-ro*b[i3]-d1*c[i3]))/length[i];

                        cb = a[i1]*a_dna[ic1] + a[i2]*a_dna[ic2] + a[i3]*a_dna[ic3];
                        if (cb > 1.0) cb = 1.0;
                        if (cb < -1.0) cb = -1.0;
                        beta_p[ic] = acos(cb);
                        sb = sin(beta_p[ic]);

			if (beta_p[ic] >= 1e-10){
                                b_m[0] = (a_dna[ic1]-cb*a[i1])/sb;
                                b_m[1] = (a_dna[ic2]-cb*a[i2])/sb;
                                b_m[2] = (a_dna[ic3]-cb*a[i3])/sb;
                                ca = b_m[0]*b[i1]+b_m[1]*b[i2]+b_m[2]*b[i3];
                                if (ca > 1.0) ca = 1.0;
                                if (ca < -1.0) ca = -1.0;
                                Ac = acos(ca);
                                f1 = a_dna[ic1]*c[i1]+a_dna[ic2]*c[i2]+a_dna[ic3]*c[i3];
                                if (f1 >= 0){
                                        alpha_p[ic] = Ac;
                                }else{
                                        alpha_p[ic] = -Ac;
                                }
                                gamma_p[ic] = -alpha_p[ic];
                                sa = sin(alpha_p[ic]);
                                sg = sin(gamma_p[ic]);
                                cg = cos(gamma_p[ic]);
                                R21 = -cg*sb;
                                R22 = cg*cb*ca-sg*sa;
                                R23 = cg*cb*sa+sg*ca;

                                b_dna[ic1] = R21*a[i1] + R22*b[i1] + R23*c[i1];
                                b_dna[ic2] = R21*a[i2] + R22*b[i2] + R23*c[i2];
                                b_dna[ic3] = R21*a[i3] + R22*b[i3] + R23*c[i3];

                                R31 = sg*sb;
                                R32 = -sg*cb*ca-cg*sa;
                                R33 = -sg*cb*sa+cg*ca;

                                c_dna[ic1] = R31*a[i1]+R32*b[i1]+R33*c[i1];
                                c_dna[ic2] = R31*a[i2]+R32*b[i2]+R33*c[i2];
                                c_dna[ic3] = R31*a[i3]+R32*b[i3]+R33*c[i3];


                        }else{
				b_dna[ic1] = b[i1];
                                b_dna[ic2] = b[i2];
                                b_dna[ic3] = b[i3];
                                c_dna[ic1] = c[i1];
                                c_dna[ic2] = c[i2];
                                c_dna[ic3] = c[i3];
                        }
                        ada = a_dna[ic1]*a[if1]+a_dna[ic2]*a[if2]+a_dna[ic3]*a[if3];
                        if (ada > 1.0) ada = 1.0;
                        if (ada < -1.0) ada = -1.0;
                        beta[i] = acos(ada);
                        sb  = sin(beta[i]);
                        if (beta[i] >= 1e-10){
                                f1 = (a[if1]*b_dna[ic1]+a[if2]*b_dna[ic2]+a[if3]*b_dna[ic3])/sb;
                        }else{
                                f1 = (b[if1]*b_dna[ic1]+b[if2]*b_dna[ic2]+b[if3]*b_dna[ic3]);
                        }
                        if (f1 > 1.0) f1 =1.0;
                        if (f1 < -1.0) f1 = -1.0;
                        Ac = acos(f1);
                        f2 = a[if1]*c_dna[ic1]+a[if2]*c_dna[ic2]+a[if3]*c_dna[ic3];
                        if (f2 >= 0){
                                alpha[i] = Ac;
                        }else{
                                alpha[i] = -Ac;
                        }

                        f1 = (b_dna[ic1]*b[if1]+b_dna[ic2]*b[if2]+b_dna[ic3]*b[if3]+c_dna[ic1]*c[if1]+c_dna[ic2]*c[if2]+c_dna[ic3]*c[if3])/(1.0+ada);
                        if (f1 > 1.0) f1 = 1.0;
                        if (f1 < -1.0) f1 = -1.0;
                        apg = acos(f1);
                        f2 = (c_dna[ic1]*b[if1]+c_dna[ic2]*b[if2]+c_dna[ic3]*b[if3]-(b_dna[ic1]*c[if1]+b_dna[ic2]*c[if2]+b_dna[ic3]*c[if3]))/(1.0+ada);
                        if (f2 >= 0.0){
                                gamma[i] = apg - alpha[i];
                        }else{
                                gamma[i] = -apg - alpha[i];
                        }

                }

        }
}


__global__ void mechanical_force_and_torque_cuda(int maxrtlh, int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p, double h, double g, double s, double* phi_o, double* force, double* torque, double* Energy, double* force_m, double* torque_m, double* Energy_m){

        int ic, count;
        int i1, i2, i3, ib1, ib2, ib3, ic1, ic2, ic3, im1;
        double c1, s1, si, co;
        double a_m[3];
        double alpha_b, beta_b, gamma_b, phi_o_b, gamma_n;
        double a_f[3], a_b[3], a_o[3], b_o[3], c_o[3];
        double mag;
        double ada, adb, adc, cda, bda, cdb, bdb, cdc, bdc;

        double torque1[3], torque2[3];
        double force_projection1[3], force_projection2[3];
        double r_f[3], r_s[3];

        double r_tmp1[3], r_tmp2[3], a_tmp1[3], a_tmp2[3], b_tmp1[3], b_tmp2[3], c_tmp1[3], c_tmp2[3];

	si = sin(theta);
        co = cos(theta);

        int column = blockIdx.x * blockDim.x + threadIdx.x;

        //Stretching

        if (column<n-1){

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

                first_coord_cuda (type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
                second_coord_cuda (type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                Force_Stretching_cuda (h,r_s,r_f,lo,force_projection1,force_projection2, Energy_m[column]);

                force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

                force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda (force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
		torque_due_to_force_cuda (force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

                torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

                torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

        __threadfence();

	//Bending
        if (column < n-1){

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

                first_coord_cuda(type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
                second_coord_cuda(type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                if (type[column]==0){
                        if (type[column+1]==0){
                                for (int xi = 0; xi<3; xi++){
                                        a_f[xi] = a[column*3+3+xi];
                                }
                        }else{
                                a_m[0] = co*a[column*3+3] + si*b[column*3+3];
                                a_m[1] = co*a[column*3+4] + si*b[column*3+4];
                                a_m[2] = co*a[column*3+5] + si*b[column*3+5];
                                for (int xi = 0; xi<3; xi++){
                                        a_f[xi] = a_m[xi];
                                }
                        }
                        if (type[column-1]==0){
                                for (int xi = 0; xi<3; xi++){
                                        a_b[xi] = a[column*3-3+xi];
                                }
                        }else{
                                for (int xi = 0; xi<3; xi++){
                                        a_b[xi] = a_dna[ic1+xi];
                                }
                        }
			for (int xi = 0; xi<3; xi++){
                                a_o[xi] = a[column*3+xi];
                        }
                        beta_b = beta[column-1];

                }else{
                        for (int xi = 0; xi<3; xi++){
                                a_f[xi] = a[column*3+3+xi];
                                a_b[xi] = a[column*3+xi];
                                a_o[xi] = a_dna[ic1+xi];
                        }
                        beta_b = beta_p[ic];
                }

                Bending_force_projection_cuda(g, beta[column], beta_b, length[column], a_f, a_b, a_o, force_projection1, force_projection2, Energy_m[column]);

                force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

                force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda(force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
                torque_due_to_force_cuda(force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

                torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

                torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

	__threadfence();

        //Twisting

        if (column < n-1){

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

		first_coord_cuda(type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
                second_coord_cuda(type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                if (type[column]==0){
                        alpha_b = alpha[column-1];
                        beta_b = beta[column-1];
                        gamma_b = gamma[column-1];
                        phi_o_b = phi_o[column-1];
                        gamma_n = gamma[column-1];
                        for (int xi = 0; xi <3; xi++){
                                b_o[xi] = b[column*3+xi];
                                c_o[xi] = c[column*3+xi];
                        }
                }else{
                        alpha_b = alpha[column];
                        beta_b = beta_p[ic];
                        gamma_b = gamma[column];
                        phi_o_b = phi_o[column];
                        gamma_n = gamma_p[ic];
                        for (int xi = 0; xi <3; xi++){
                                b_o[xi] = b_dna[ic1+xi];
                                c_o[xi] = c_dna[ic1+xi];
                        }
                }

                Twisting_force_projection_cuda(s, alpha[column], beta[column], gamma[column], phi_o[column],  length[column], alpha_b, beta_b, gamma_b, phi_o_b, gamma_n, b_o, c_o, force_projection1, force_projection2, Energy_m[column]);

		force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

                force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda(force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
                torque_due_to_force_cuda(force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

                torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

                torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

        __threadfence();

	//Mechanical Torques

        if (column < n-1){

                im1 = column-1;
                i1 = column*3;
                i2 = i1+1;
                i3 = i2+1;
                ib1 = i1-3;
                ib2 = ib1+1;
                ib3 = ib2+1;

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                if (type[column]==0){
                        torque[i1] = s*(alpha[column]+gamma[column]+phi_o[column]-alpha[im1]-gamma[im1]-phi_o[im1]);
                        torque[i2] = 0.0;
                        torque[i3] = 0.0;
		}else{
                        ada = a_dna[ic1]*a[i1] + a_dna[ic2]*a[i2] + a_dna[ic3]*a[i3];
                        adb = a_dna[ic1]*b[i1] + a_dna[ic2]*b[i2] + a_dna[ic3]*b[i3];
                        adc = a_dna[ic1]*c[i1] + a_dna[ic2]*c[i2] + a_dna[ic3]*c[i3];

                        mag = s*(alpha[column]+gamma[column]-phi_o[column]);
                        torque[i1] = torque[i1] + mag*ada;
                        torque[i2] = torque[i2] + mag*adb;
                        torque[i3] = torque[i3] + mag*adc;
                        if (column > 0){
                                mag = -s*(alpha[im1]+gamma[im1]-phi_o[im1]);
                                torque[i1] = torque[i1] + mag*co;
                                torque[i2] = torque[i2] + mag*si;
                                torque[i3] = torque[i3] + 0.0;
                        }

                        //Extra Bending torque

                        torque[i2] = torque[i2] - g*beta_p[ic]*adc/sin(beta_p[ic]);
                        torque[i3] = torque[i3] + g*beta_p[ic]*adb/sin(beta_p[ic]);

                        if (column > 0){
                                ada = a[ib1]*a[i1] + a[ib2]*a[i2] + a[ib3]*a[i3];
                                adb = a[ib1]*b[i1] + a[ib2]*b[i2] + a[ib3]*b[i3];
                                adc = a[ib1]*c[i1] + a[ib2]*c[i2] + a[ib3]*c[i3];

                                torque[i1] = torque[i1] + g*beta[im1]*(si*adc)/sin(beta[im1]);
                                torque[i2] = torque[i2] - g*beta[im1]*(co*adc)/sin(beta[im1]);
                                torque[i3] = torque[i3] + g*beta[im1]*(co*adb-si*ada)/sin(beta[im1]);
                        }

			//Extra Twisting torque

                        s1 = sin(alpha_p[ic]);
                        c1 = cos(alpha_p[ic]);
                        mag = s*(alpha[column]+gamma[column]-phi_o[column])*tan(0.5*beta_p[ic]);
                        cda = c_dna[ic1]*a[i1] + c_dna[ic2]*a[i2] + c_dna[ic3]*a[i3];
                        bda = b_dna[ic1]*a[i1] + b_dna[ic2]*a[i2] + b_dna[ic3]*a[i3];
                        cdb = c_dna[ic1]*b[i1] + c_dna[ic2]*b[i2] + c_dna[ic3]*b[i3];
                        bdb = b_dna[ic1]*b[i1] + b_dna[ic2]*b[i2] + b_dna[ic3]*b[i3];
                        cdc = c_dna[ic1]*c[i1] + c_dna[ic2]*c[i2] + c_dna[ic3]*c[i3];
                        bdc = b_dna[ic1]*c[i1] + b_dna[ic2]*c[i2] + b_dna[ic3]*c[i3];

                        torque[i1] = torque[i1] - mag*(s1*cda + c1*bda);
                        torque[i2] = torque[i2] - mag*(s1*cdb + c1*bdb);
                        torque[i3] = torque[i3] - mag*(s1*cdc + c1*bdc);

                        if (column > 0){
                                s1 = sin(gamma[im1]);
                                c1 = cos(gamma[im1]);
                                mag = s*(alpha[im1]+gamma[im1]-phi_o[im1])*tan(0.5*beta[im1]);
                                cda = 0.0;
                                bda = -si;
                                cdb = 0.0;
                                bdb = co;
                                cdc = 1.0;
                                bdc = 0.0;

                                torque[i1] = torque[i1] - mag*(s1*cda - c1*bda);
                                torque[i2] = torque[i2] - mag*(s1*cdb - c1*bdb);
                                torque[i3] = torque[i3] - mag*(s1*cdc - c1*bdc);
                        }

                }

        }

	__threadfence();

        //Additional torque for last bead

        if (column==0){

                torque[n3-3] = -s*(alpha[n-2]+gamma[n-2]-phi_o[n-2]);
                torque[n3-2] = 0.0;
                torque[n3-1] = 0.0;

        }


}


__global__ void Electrostatic_and_Excluded_volume_force_cuda(int maxrtlh, int n, int n3, int n_c, int nc3, int* type, double* r, double* a, double* b, double* c, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, int Nq, int Nq3, double* core_pos, double* core_q, double* force, double* torque, double* Energy, double* force_m, double* torque_m, double* Energy_m){

        double ql_ql, dist;
        int k, l, ch;
        int i1, i2, i3, j1, j2, j3, k1, k2, k3, l1, l2, l3;
        double mi, Rcut, temp, temp1, temp2;
        double z[3];
        double fa, fb, fc;
        double g1, s1, s2;


        Rcut = 25.0;
        ql_ql = q_l*q_l;

        // Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;

                if (j >=  i+1 and j < n){
                        j1 = j*3;
                        j2 = j1+1;
                        j3 = j2+1;

                        dist = (r[j1]-r[i1])*(r[j1]-r[i1])+(r[j2]-r[i2])*(r[j2]-r[i2])+(r[j3]-r[i3])*(r[j3]-r[i3]);
                        dist = sqrt(dist);

			ch = 1;
                        if (dist > Rcut) ch=0;
                        if (type[i+1] != 0) ch=0;
                        if (i > 0 and type[i-1] != 0) ch=0;
                        if (j < n-1){
                                if (type[j+1] != 0) ch=0;
                        }
                        if (type[j-1] != 0) ch=0;


                        if (ch == 1){
                                if (type[i] == 0){
                                        if (type[j] == 0){
                                                if (abs(i-j) > 1){
                                                        mi = 1.0/dist;
                                                        z[0] = mi*(r[i1]-r[j1]);
                                                        z[1] = mi*(r[i2]-r[j2]);
                                                        z[2] = mi*(r[i3]-r[j3]);

                                                        temp = -debyell*dist;

                                                        g1 = k_e*ql_ql*exp(temp)*(debyell*dist+1.0)/(dist*dist);

                                                        Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*ql_ql/dist)*exp(temp);
                                                        if (dist <= 8){
                                                                s1 = sigma_DNA_DNA;
                                                                s2 = sigma_DNA_DNA;
                                                                temp1 = s1/dist;
                                                                temp2 = s2/dist;
                                                                g1 = g1 + k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));
                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));
                                                        }

                                                        force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                        force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                        force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                }
                                        }else{
						//DNA-Core interaction

                                                for (k=0;k<Nq;k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        z[0] = (r[i1]-(r[j1]+a[j1]*core_pos[k1]+b[j1]*core_pos[k2]+c[j1]*core_pos[k3]));
                                                        z[1] = (r[i2]-(r[j2]+a[j2]*core_pos[k1]+b[j2]*core_pos[k2]+c[j2]*core_pos[k3]));
                                                        z[2] = (r[i3]-(r[j3]+a[j3]*core_pos[k1]+b[j3]*core_pos[k2]+c[j3]*core_pos[k3]));
                                                        dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
                                                        dist = sqrt(dist);
                                                        mi = 1.0/dist;
                                                        z[0] = mi*z[0];
                                                        z[1] = mi*z[1];
                                                        z[2] = mi*z[2];

                                                        if (abs(i-j)>1){

                                                                temp = -debyell*dist;
                                                                g1 = k_e*q_l*core_q[k]*exp(temp)*(debye*dist+1.0)/(dist*dist);
                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*q_l*core_q[k]/dist)*exp(temp);

                                                                force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[k1] - fa*core_pos[k2];



                                                        }

							// Excluded Volume force

                                                        if (dist <= 8.0 and core_q[k]>0){
                                                                s1 = sigma_DNA_Core;
                                                                s2 = sigma_DNA_Core;
                                                                temp1 = s1/dist;
                                                                temp2 = s2/dist;
                                                                g1 = k_ex*((12.0/s1)*pow((temp1),13)-(6.0/s2)*pow((temp2),7));
                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));


                                                                force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                        }

                                                }
                                        }

                                }else{
					//Core-DNA interaction
                                        if (type[j] == 0){
                                                for (k=0;k<Nq;k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        z[0] = (-r[j1]+(r[i1]+a[i1]*core_pos[k1]+b[i1]*core_pos[k2]+c[i1]*core_pos[k3]));
                                                        z[1] = (-r[j2]+(r[i2]+a[i2]*core_pos[k1]+b[i2]*core_pos[k2]+c[i2]*core_pos[k3]));
                                                        z[2] = (-r[j3]+(r[i3]+a[i3]*core_pos[k1]+b[i3]*core_pos[k2]+c[i3]*core_pos[k3]));
                                                        dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
                                                        dist = sqrt(dist);

                                                        mi = 1.0/dist;
                                                        z[0] = mi*z[0];
                                                        z[1] = mi*z[1];
                                                        z[2] = mi*z[2];
                                                        if (abs(i-j) > 1){
                                                                temp = -debye*dist;
                                                                g1 = k_e*q_l*core_q[k]*exp(temp)*(debye*dist+1.0)/(dist*dist);
                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*q_l*core_q[k]/dist)*exp(temp);

                                                                force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

                                                                torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                        }

							//Excluded Volume force
                                                        if (dist <= 8.0 and core_q[k]>0){
                                                                s1 = sigma_DNA_Core;
                                                                s2 = sigma_DNA_Core;
                                                                temp1 = s1/dist;
                                                                temp2 = s2/dist;

                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));
                                                                g1 = k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));

                                                                force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

                                                                torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                        }
                                                }
					}else{//Core-Core interaction

                                                for (k = 0; k < Nq; k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        for (l = 0; l < Nq; l++){
                                                                l1 = 3*l;
                                                                l2 = l1+1;
                                                                l3 = l2+1;
                                                                z[0] = (r[i1]+a[i1]*core_pos[k1]+b[i1]*core_pos[k2]+c[i1]*core_pos[k3] - (r[j1]+a[j1]*core_pos[l1]+b[j1]*core_pos[l2]+c[j1]*core_pos[l3]));
                                                                z[1] = (r[i2]+a[i2]*core_pos[k1]+b[i2]*core_pos[k2]+c[i2]*core_pos[k3] - (r[j2]+a[j2]*core_pos[l1]+b[j2]*core_pos[l2]+c[j2]*core_pos[l3]));
                                                                z[2] = (r[i3]+a[i3]*core_pos[k1]+b[i3]*core_pos[k2]+c[i3]*core_pos[k3] - (r[j3]+a[j3]*core_pos[l1]+b[j3]*core_pos[l2]+c[j3]*core_pos[l3]));
                                                                dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
                                                                dist = sqrt(dist);

                                                                mi = 1.0/dist;
                                                                z[0] = mi*z[0];
                                                                z[1] = mi*z[1];
                                                                z[2] = mi*z[2];

                                                                temp = -debye*dist;
                                                                g1 = k_e*core_q[k]*core_q[l]*exp(temp)*(debye*dist+1.0)/(dist*dist);
                                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*core_q[l]*core_q[k]/dist)*exp(temp);

                                                                force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

                                                                torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                                fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[l2] - fb*core_pos[l3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[l3] - fc*core_pos[l1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[l1] - fa*core_pos[l2];
								//Excluded Volume force
                                                                if (dist <= 8.0){
                                                                        s1 = sigma_Core_Core;
                                                                        s2 = sigma_Core_Core;
                                                                        temp1 = s1/dist;
                                                                        temp2 = s2/dist;
                                                                        g1 = k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));

                                                                        Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));

                                                                        force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                        force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                        force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];


                                                                        //torque due to force
                                                                        fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                        fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                        fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

                                                                        torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                        torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                        torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                                        fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                        fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                        fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[l2] - fb*core_pos[l3];
                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[l3] - fc*core_pos[l1];
                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[l1] - fa*core_pos[l2];

                                                                }

                                                        }
                                                }

                                        }
                                }

                        }


                        }






        }

}




__global__ void extra_force_cuda(int n, int n3, int ex_n, int* type, double* r, double h, double* force, double* Energy, double* force_m, double* Energy_m, int* ex_force_m){

	double force_projection1[3], force_projection2[3];
        double r_f[3], r_s[3];
	int i,j;

	// Calculate Row and Column
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index<ex_n){
		i=ex_force_m[index*2]-1;
		j=ex_force_m[index*2+1]-1;
		for (int xi=0;xi<3;xi++){
                        r_f[xi]=r[i*3+xi];
                       	r_s[xi]=r[j*3+xi];
                }
		Force_Stretching_cuda (h*((double)j-(double)i)/100.0,r_f,r_s,lo+((double)j-(double)i)/10.0,force_projection1,force_projection2, Energy_m[index]);
		force_m[i*2*n3+j*3] += force_projection1[0];
	        force_m[i*2*n3+j*3+1] += force_projection1[1];
        	force_m[i*2*n3+j*3+2] += force_projection1[2];

		force_m[j*2*n3+n3+i*3] += force_projection2[0];
                force_m[j*2*n3+n3+i*3+1] += force_projection2[1];
                force_m[j*2*n3+n3+i*3+2] += force_projection2[2];

	}
	
}

__global__ void init_force_torque_m(int maxrtlh, double* force_m, double* torque_m, double* Energy_m){

        // Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < maxrtlh*2 && j < maxrtlh*3){
                force_m[i*maxrtlh*3+j] = 0.0;
                torque_m[i*maxrtlh*3+j] = 0.0;
        }

        if (i < maxrtlh && j< maxrtlh){
                Energy_m[i*maxrtlh+j]=0.0;
        }

}

__global__ void step_init(int n, double* force, double* force_n, double* torque, double* torque_n,  double* Energy, double* r_all, double* r){

        int i = blockIdx.x * blockDim.x + threadIdx.x;

        Energy[0] = 0.0;
        if (i < n*3){
                force[i]=0;
                force_n[i]=0;
                torque[i]=0;
                torque_n[i]=0;
        }

        if (i<n){
                r_all[i*3] = r[i*3];
                r_all[i*3+1] = r[i*3+1];
                r_all[i*3+2] = r[i*3+2];
        }

}

__global__ void force_reduction(int n, int maxrtlh, int maxrtlh3, double* force_m, double* force, double* torque_m, double* torque){

        int column = blockIdx.x * blockDim.x + threadIdx.x;
        int index1, index2;

        if (column < maxrtlh){
                for (int i=0; i<maxrtlh; i++){
                        index1 = column*2*maxrtlh3+i*3;
                        index2 = column*2*maxrtlh3+maxrtlh3+i*3;
                        if (column<n){
                                force[column*3] += force_m[index1] + force_m[index2];
                                force[column*3+1] += force_m[index1+1] + force_m[index2+1];
                                force[column*3+2] += force_m[index1+2] + force_m[index2+2];
                        }
                        if (column<n){
                                torque[column*3] += torque_m[index1] + torque_m[index2];
                                torque[column*3+1] += torque_m[index1+1] + torque_m[index2+1];
                                torque[column*3+2] += torque_m[index1+2] + torque_m[index2+2];
                        }
                }
        }

}

__global__ void Energy_reduction(int maxrtlh, double* Energy, double* Energy_m){
        int column = blockIdx.x * blockDim.x + threadIdx.x;

        if (column< maxrtlh){
                for (int i=1; i<maxrtlh; i++){
                        Energy_m[column] += Energy_m[i*maxrtlh+column];
                }
        }

        __threadfence();

        if (column==0){
                for (int i=0; i<maxrtlh; i++){
                        Energy[0] += Energy_m[i];
                }
        }


}

__global__ void force_torque_tmp(int n3, double* force, double* torque, double* force_n, double* torque_n, double* force_tmp, double* torque_tmp){

        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < n3){
                force_tmp[i] = 0.5*(force[i]+force_n[i]);
                torque_tmp[i] = 0.5*(torque[i]+torque_n[i]);
        }

}

__global__ void final_updates(int n3, double* r, double* r_n, double* a, double* a_n, double* b, double* b_n, double* c, double* c_n){

        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < n3){
                r[i] = r_n[i];
                a[i] = a_n[i];
                b[i] = b_n[i];
                c[i] = c_n[i];
        }

}

extern "C++" void cuda_application_init_D_Chol(int n3){


        size_t bytes_D = n3*n3*sizeof(double);
        size_t bytes_Chol = n3*n3*sizeof(double);

        hipMalloc(&d_D, bytes_D);
        hipMalloc(&d_Chol, bytes_Chol);

}

extern "C++" void cuda_application_init_data(int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p, double h, double g, double s, double* phi_o, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, int Nq, int Nq3, double* core_pos, double* core_q, double* force, double* torque, double* Energy, double* r_all, double* rad_all, int ex_n, int* ex_force_m){

	int n_D = n;
        int n_D3 = n_D*3;

        size_t bytes = sizeof(double);
        size_t bytes_ni = n*sizeof(int);
        size_t bytes_nd = n*sizeof(double);
        size_t bytes_n3 = n3*sizeof(double);
        size_t bytes_nc = n_c*sizeof(double);
        size_t bytes_nc3 = nc3*sizeof(double);
        size_t bytes_Nq = Nq*sizeof(double);
        size_t bytes_Nq3 = Nq3*sizeof(double);

	size_t bytes_n_D = n_D*sizeof(double);
        size_t bytes_n_D3 = n_D3*sizeof(double);

	size_t bytes_ex_n = 2*ex_n*sizeof(int);

	int maxrtlh;

	maxrtlh = n;

	size_t bytes_r_t_lh_m = maxrtlh*maxrtlh*6*sizeof(double);

        size_t bytes_r_t_lh_m_E = maxrtlh*maxrtlh*sizeof(double);

	hipMalloc(&d_Energy, bytes);
        hipMalloc(&d_type, bytes_ni);
        hipMalloc(&d_r, bytes_n3);
        hipMalloc(&d_a, bytes_n3);
        hipMalloc(&d_b, bytes_n3);
        hipMalloc(&d_c, bytes_n3);
        hipMalloc(&d_alpha, bytes_nd);
        hipMalloc(&d_beta, bytes_nd);
        hipMalloc(&d_gamma, bytes_nd);
        hipMalloc(&d_length, bytes_nd);
        hipMalloc(&d_a_dna, bytes_nc3);
        hipMalloc(&d_b_dna, bytes_nc3);
        hipMalloc(&d_c_dna, bytes_nc3);
        hipMalloc(&d_alpha_p, bytes_nc);
        hipMalloc(&d_beta_p, bytes_nc);
        hipMalloc(&d_gamma_p, bytes_nc);
        hipMalloc(&d_phi_o, bytes_nd);
        hipMalloc(&d_core_pos, bytes_Nq3);
        hipMalloc(&d_core_q, bytes_Nq);
        hipMalloc(&d_force, bytes_n3);
        hipMalloc(&d_torque, bytes_n3);
        hipMalloc(&d_force_n, bytes_n3);
        hipMalloc(&d_torque_n, bytes_n3);

        hipMalloc(&d_force_tmp, bytes_n3);
        hipMalloc(&d_torque_tmp, bytes_n3);

        hipMalloc(&d_r_all, bytes_n_D3);
        hipMalloc(&d_rad_all, bytes_n_D);
	hipMalloc(&d_d_theta, bytes_n3);
        hipMalloc(&d_rd, bytes_n_D3);

        hipMalloc(&d_r_n, bytes_n3);
        hipMalloc(&d_a_n, bytes_n3);
        hipMalloc(&d_b_n, bytes_n3);
        hipMalloc(&d_c_n, bytes_n3);
        hipMalloc(&d_alpha_n, bytes_nd);
        hipMalloc(&d_beta_n, bytes_nd);
        hipMalloc(&d_gamma_n, bytes_nd);
        hipMalloc(&d_length_n, bytes_nd);
        hipMalloc(&d_a_dna_n, bytes_nc3);
        hipMalloc(&d_b_dna_n, bytes_nc3);
        hipMalloc(&d_c_dna_n, bytes_nc3);
        hipMalloc(&d_alpha_p_n, bytes_nc);
        hipMalloc(&d_beta_p_n, bytes_nc);
        hipMalloc(&d_gamma_p_n, bytes_nc);

	hipMalloc(&d_force_m, bytes_r_t_lh_m);
        hipMalloc(&d_torque_m, bytes_r_t_lh_m);

	hipMalloc(&d_Energy_m, bytes_r_t_lh_m_E);


	hipMalloc(&d_ex_force_m, bytes_ex_n);

	//Copy data to the device

        hipMemcpy(d_Energy, Energy, bytes,hipMemcpyHostToDevice);
        hipMemcpy(d_type, type, bytes_ni, hipMemcpyHostToDevice);
        hipMemcpy(d_r, r, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_a, a, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_alpha, alpha, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_beta, beta, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_gamma, gamma, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_length, length, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_a_dna, a_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_b_dna, b_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_c_dna, c_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_alpha_p, alpha_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_p, beta_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_gamma_p, gamma_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_phi_o, phi_o, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_core_pos, core_pos, bytes_Nq3, hipMemcpyHostToDevice);
        hipMemcpy(d_core_q, core_q, bytes_Nq, hipMemcpyHostToDevice);
        hipMemcpy(d_force, force, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_torque, torque, bytes_n3, hipMemcpyHostToDevice);

	hipMemcpy(d_r_all, r_all, bytes_n_D3, hipMemcpyHostToDevice);
        hipMemcpy(d_rad_all, rad_all, bytes_n_D, hipMemcpyHostToDevice);

	hipMemcpy(d_ex_force_m, ex_force_m, bytes_ex_n, hipMemcpyHostToDevice);

}

extern "C++" void main_cuda(int n_c, int nc3, int ex_n, int step, int number_of_steps, double time_step, double del, int frequency_RP, int frequency_of_sampling, double h, double g, double s, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, int Nq, int Nq3, int n, int n3, double a1, double a2, double s2dt, double* rr, double* p, double* Energy, double* h_r, double* h_a, double* h_b, double* h_c, double* h_rad_all){

	int maxrtlh;
	maxrtlh=n;
	int n_D, n_D3;
	n_D=n;
	n_D3=n3;

	size_t bytes = sizeof(double);

	size_t bytes_n_D = n_D*sizeof(double);
        size_t bytes_n_D3 = n_D3*sizeof(double);

	hipMalloc(&d_r_all, bytes_n_D3);
        hipMalloc(&d_rad_all, bytes_n_D);

	hipMemcpy(d_rad_all, h_rad_all, bytes_n_D, hipMemcpyHostToDevice);

	int num_thread_rtlh3 = 64;
        int num_block_rtlh3 = (maxrtlh*3 + num_thread_rtlh3 -1) / num_thread_rtlh3;

        step_init<<<num_block_rtlh3, num_thread_rtlh3>>>(n, d_force, d_force_n, d_torque, d_torque_n, d_Energy, d_r_all, d_r);

	hipFree(d_D);
        hipFree(d_Chol);

	size_t bytes_D = n_D3*n_D3*sizeof(double);

	hipMalloc(&d_D, bytes_D);
        hipMalloc(&d_Chol, bytes_D);

	hipsolverHandle_t solver_handle;
        hipsolverDnCreate(&solver_handle);

        int work_size = 0;
        int *devInfo;
        hipMalloc(&devInfo, sizeof(int));

	int threads_per_block_D = 32;
        dim3 block_size_D(threads_per_block_D, threads_per_block_D);
        dim3 grid_size_D(n_D3 / block_size_D.x + 1 , n_D3 / block_size_D.y + 1 );

        Diffusion_Tensor_CUDA <<<grid_size_D, block_size_D>>> (n_D, n_D3, d_r_all, a1, a2, d_rad_all, d_D, d_Chol);

        hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_UPPER, n_D3, d_Chol, n_D3, &work_size);

	double *work;
        hipMalloc(&work, work_size * sizeof(double));
        hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, n_D3, d_Chol, n_D3, work, work_size, devInfo);
        Cholesky_Decomposition_mod <<<grid_size_D, block_size_D>>> (n_D3, d_Chol);
        hipFree(devInfo);
        hipFree(work);

	hipsolverDnDestroy(solver_handle);

	double* d_p;
        double* d_rr;

        size_t bytes_p = n_D3*sizeof(double);
        size_t bytes_rr = n3*sizeof(double);

        hipMalloc(&d_p, bytes_p);
        hipMalloc(&d_rr, bytes_rr);

        hipMemcpy(d_p, p, bytes_p, hipMemcpyHostToDevice);
        hipMemcpy(d_rr, rr, bytes_rr, hipMemcpyHostToDevice);

	int num_thread_rd = 32;
        int num_block_rd = (n_D3 + num_thread_rd -1) / num_thread_rd;

        rd_cal <<<num_block_rd, num_thread_rd>>> (n_D3, d_rd, d_Chol, d_p, s2dt);

	//Force and torque calculation

        int threads_per_block_rtlh = 16;
        dim3 block_size_rtlh(threads_per_block_rtlh, threads_per_block_rtlh);
        dim3 grid_size_rtlh(maxrtlh*3 / block_size_rtlh.x + 1 , maxrtlh*3 / block_size_rtlh.y + 1 );

        init_force_torque_m<<<grid_size_rtlh, block_size_rtlh>>>(maxrtlh, d_force_m, d_torque_m, d_Energy_m);

	int num_thread = 64;
        int num_block = (n + num_thread -1) / num_thread;

        mechanical_force_and_torque_cuda <<<num_block, num_thread>>> (maxrtlh, n_c, nc3, n, n3, d_type, d_r, d_a, d_b, d_c, d_alpha, d_beta, d_gamma, d_length, d_a_dna, d_b_dna, d_c_dna, d_alpha_p, d_beta_p, d_gamma_p, h, g, s, d_phi_o, d_force, d_torque, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	int threads_per_block = 16;
        dim3 block_size(threads_per_block, threads_per_block);
        dim3 grid_size(n / block_size.x + 1 , n / block_size.y + 1 );

//        Electrostatic_and_Excluded_volume_force_cuda <<<grid_size, block_size>>> (maxrtlh, n, n3, n_c, nc3, d_type, d_r, d_a, d_b, d_c, debyell, debye, q_l, k_e, k_ex, k_h1, sigma_DNA_DNA, sigma_DNA_Core, sigma_Core_Core, Nq, Nq3, d_core_pos, d_core_q, d_force, d_torque, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	int num_thread_ex = 64;
	int num_block_ex = (ex_n+num_thread-1) / num_thread_ex;
	extra_force_cuda <<<num_block_ex, num_thread_ex>>> (n, n3, ex_n, d_type, d_r, h/20, d_force, d_Energy, d_force_m, d_Energy_m, d_ex_force_m);

	int num_thread_rtlh = 64;
        int num_block_rtlh = (maxrtlh + num_thread_rtlh -1) / num_thread_rtlh;

	force_reduction <<<num_block_rtlh, num_thread_rtlh>>> (n, maxrtlh, maxrtlh*3, d_force_m, d_force, d_torque_m, d_torque);

	Energy_reduction  <<<num_block_rtlh, num_thread_rtlh>>> (maxrtlh, d_Energy, d_Energy_m);

	hipMemcpy(Energy, d_Energy, bytes, hipMemcpyDeviceToHost);

	rotation_cal <<<num_block, num_thread>>> (n, d_d_theta, d_type, time_step, d_torque, d_rr, d_a, d_b, d_c, d_a_n, d_b_n, d_c_n);

	translation_cal <<<num_block_rd, num_thread_rd>>> (n_D3, n3, d_r, d_r_n, d_rd, del, d_force, d_D);

	update_Euler_Angle_cuda <<<num_block, num_thread>>> (n_c, nc3, n, n3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, d_alpha_n, d_beta_n, d_gamma_n, d_length_n, d_a_dna_n, d_b_dna_n, d_c_dna_n, d_alpha_p_n, d_beta_p_n, d_gamma_p_n);

	init_force_torque_m<<<grid_size_rtlh, block_size_rtlh>>>(maxrtlh, d_force_m, d_torque_m, d_Energy_m);

	mechanical_force_and_torque_cuda <<<num_block, num_thread>>> (maxrtlh, n_c, nc3, n, n3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, d_alpha_n, d_beta_n, d_gamma_n, d_length_n, d_a_dna_n, d_b_dna_n, d_c_dna_n, d_alpha_p_n, d_beta_p_n, d_gamma_p_n, h, g, s, d_phi_o, d_force_n, d_torque_n, d_Energy, d_force_m, d_torque_m, d_Energy_m);

//	Electrostatic_and_Excluded_volume_force_cuda <<<grid_size, block_size>>> (maxrtlh, n, n3, n_c, nc3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, debyell, debye, q_l, k_e, k_ex, k_h1, sigma_DNA_DNA, sigma_DNA_Core, sigma_Core_Core, Nq, Nq3, d_core_pos, d_core_q, d_force_n, d_torque_n, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	extra_force_cuda <<<num_block_ex, num_thread_ex>>> (n, n3, ex_n, d_type, d_r_n, h/20, d_force_n, d_Energy, d_force_m, d_Energy_m, d_ex_force_m);

	force_reduction <<<num_block_rtlh, num_thread_rtlh>>> (n, maxrtlh, maxrtlh*3, d_force_m, d_force_n, d_torque_m, d_torque_n);

	force_torque_tmp<<<num_block_rtlh3, num_thread_rtlh3>>>(n3, d_force, d_torque, d_force_n, d_torque_n, d_force_tmp, d_torque_tmp);

	rotation_cal <<<num_block, num_thread>>> (n, d_d_theta, d_type, time_step, d_torque_tmp, d_rr, d_a, d_b, d_c, d_a_n, d_b_n, d_c_n);

	translation_cal <<<num_block_rd, num_thread_rd>>> (n_D3, n3, d_r, d_r_n, d_rd, del, d_force_tmp, d_D);

	final_updates <<<num_block_rtlh3, num_thread_rtlh3>>> (n3, d_r, d_r_n, d_a, d_a_n, d_b, d_b_n, d_c, d_c_n);

        update_Euler_Angle_cuda <<<num_block, num_thread>>> (n_c, nc3, n, n3, d_type, d_r, d_a, d_b, d_c, d_alpha, d_beta, d_gamma, d_length, d_a_dna, d_b_dna, d_c_dna, d_alpha_p, d_beta_p, d_gamma_p);

	if (step%frequency_of_sampling == 0 or step == number_of_steps-1){
                size_t bytes_r = n3*sizeof(double);

                hipMemcpy(h_r, d_r, bytes_r, hipMemcpyDeviceToHost);
                hipMemcpy(h_a, d_a, bytes_r, hipMemcpyDeviceToHost);
                hipMemcpy(h_b, d_b, bytes_r, hipMemcpyDeviceToHost);
                hipMemcpy(h_c, d_c, bytes_r, hipMemcpyDeviceToHost);
        }



        hipFree(d_p);
        hipFree(d_rr);

}



extern "C++" void free_all(){

	hipFree(d_Energy);
        hipFree(d_type);
        hipFree(d_r);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipFree(d_alpha);
        hipFree(d_beta);
        hipFree(d_gamma);
        hipFree(d_length);
        hipFree(d_a_dna);
        hipFree(d_b_dna);
        hipFree(d_c_dna);
        hipFree(d_alpha_p);
        hipFree(d_beta_p);
        hipFree(d_gamma_p);
        hipFree(d_phi_o);
        hipFree(d_force);
        hipFree(d_torque);
        hipFree(d_core_pos);
        hipFree(d_core_q);

	hipFree(d_r_n);
        hipFree(d_a_n);
        hipFree(d_b_n);
        hipFree(d_c_n);
        hipFree(d_alpha_n);
        hipFree(d_beta_n);
        hipFree(d_gamma_n);
        hipFree(d_length_n);
        hipFree(d_a_dna_n);
        hipFree(d_b_dna_n);
        hipFree(d_c_dna_n);
        hipFree(d_alpha_p_n);
        hipFree(d_beta_p_n);
        hipFree(d_gamma_p_n);

        hipFree(d_force_n);
        hipFree(d_torque_n);
        hipFree(d_force_tmp);
        hipFree(d_torque_tmp);

        hipFree(d_r_all);
        hipFree(d_rad_all);
        hipFree(d_d_theta);
        hipFree(d_rd);

	hipFree(d_force_m);
        hipFree(d_torque_m);

	hipFree(d_D);
        hipFree(d_Chol);

        hipFree(d_Energy_m);

	hipFree(d_ex_force_m);

}
